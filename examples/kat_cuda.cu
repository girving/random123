#include "hip/hip_runtime.h"
#include "util_cuda.h"
#include "kat.c"

#define KAT_KERNEL __global__
#define KAT_GLOBAL
#define KAT_UINT size_t
#include "kat_dev_execute.c"

void host_execute_tests(kat_instance *tests_host, size_t ntests){
    CUDAInfo *infop;
    kat_instance *tests_dev;
    size_t tests_sz;

    infop = cuda_init(NULL);

    tests_sz = ntests * sizeof(tests_host[0]);
    CHECKCALL(hipMalloc(&tests_dev, tests_sz));
    CHECKCALL(hipMemcpy(tests_dev, tests_host, tests_sz, hipMemcpyHostToDevice));

    printf("starting %zu tests on 1 blocks with 1 threads/block\n", ntests);
    fflush(stdout);

    // TO DO:  call this with parallelism, <<<infop->blocks_per_grid, infop->threads_per_block>>>
    // and then insure that each of the threads got the same result.
    dev_execute_tests<<<1, 1>>>(tests_dev, ntests);

    CHECKCALL(hipDeviceSynchronize());
    CHECKCALL(hipMemcpy(tests_host, tests_dev, tests_sz, hipMemcpyDeviceToHost));
    cuda_done(infop);
}

